#include "hip/hip_runtime.h"
/* Name:
 * ID:
 * Email:
 */

/*
 * This is code skeleton for COMP5112-17Spring assignment4
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_dijkstra cuda_dijkstra_skeleton.cu
 * Run: ./cuda_dijkstra -n <number of threads> -i <input file>,
 * you will find the output in 'output.txt' file
 *
 *  by Lipeng WANG, 5th Apr 2017
 */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <vector>
#include <climits>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <time.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;
using std::vector;
using std::ceil;
using std::memcpy;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
 {                                                                              \
     const hipError_t error = call;                                            \
     if (error != hipSuccess)                                                  \
     {                                                                          \
         fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
         fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                 hipGetErrorString(error));                                    \
         exit(1);                                                               \
     }                                                                          \
 }

/*
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and one matrix dimension convert(2D->1D) function
 */
namespace utils {
    int num_threads; //number of thread
    int N; //number of vertices
    int *mat; // the adjacency matrix

    string filename; // input file name
    string outputfile; //output file name, default: 'output.txt'

    void print_usage() {
        cout << "Usage:\n" << "\tcuda_dijkstra -n <number of threads per block> -i <input file>" << endl;
        exit(0);
    }

    int parse_args(int argc, char **argv) {
        filename = "";
        outputfile = "output.txt";
        num_threads = 0;

        int opt;
        if (argc < 2) {
            print_usage();
        }
        while ((opt = getopt(argc, argv, "n:i:o:h")) != EOF) {
            switch (opt) {
                case 'n':
                    num_threads = atoi(optarg);
                    break;
                case 'i':
                    filename = optarg;
                    break;
                case 'o':
                    outputfile = optarg;
                    break;
                case 'h':
                case '?':
                default:
                    print_usage();
            }
        }
        if (filename.length() == 0 || num_threads == 0)
            print_usage();
        return 0;
    }

    /*
     * convert 2-dimension coordinate to 1-dimension
     */
    int convert_dimension_2D_1D(int x, int y) {
        return x * N + y;
    }

    int read_file(string filename) {
        std::ifstream inputf(filename, std::ifstream::in);
        inputf >> N;
        assert(N < (1024 * 1024 *
                    20)); // input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
        mat = (int *) malloc(N * N * sizeof(int));
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++) {
                inputf >> mat[convert_dimension_2D_1D(i, j)];
            }

        return 0;
    }

    string format_path(int i, int *pred) {
        string out("");
        int current_vertex = i;
        while (current_vertex != 0) {
            string s = std::to_string(current_vertex);
            std::reverse(s.begin(), s.end());
            out = out + s + ">-";
            current_vertex = pred[current_vertex];
        }
        out = out + std::to_string(0);
        std::reverse(out.begin(), out.end());
        return out;
    }

    int print_result(int *dist, int *pred) {
        std::ofstream outputf(outputfile, std::ofstream::out);
        outputf << dist[0];
        for (int i = 1; i < N; i++) {
            outputf << " " << dist[i];
        }
        for (int i = 0; i < N; i++) {
            outputf << "\n";
            if (dist[i] >= 1000000) {
                outputf << "NO PATH";
            } else {
                outputf << format_path(i, pred);
            }
        }
        outputf << endl;
        return 0;
    }
}//namespace utils


//------You may add helper functions and global variables here------

/*
 * function: find the local minimum for each block and store them to d_local_min and d_local_min_index
 * parameters: N: input size, *d_visit: array to record which vertex has been visited, *d_all_dist: array to store the distance,
 *        *d_local_min: array to store the local minimum value for each block, *d_local_min_index: array to store the local minimum index for each block
 */
__global__ void FindLocalMin(int N, int *d_visit, int *d_all_dist, int *d_local_min, int *d_local_min_index) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    __shared__ int local_mins[1024];
    __shared__ int local_mins_index[1024];

    int thread_min = INF;
    int thread_min_index = -1;

    for (int i = tid; i < N; i += elementSkip) {
        int global_vertices_id = i;

        if (d_visit[global_vertices_id] == 0) {
            if (d_all_dist[global_vertices_id] < thread_min) {
                thread_min = d_all_dist[global_vertices_id];
                thread_min_index = global_vertices_id;
            }
        }
    }
    local_mins[threadIdx.x] = thread_min;
    local_mins_index[threadIdx.x] = thread_min_index;
    __syncthreads();//make sure that every threads have stored their minimums to shared memory

    for (int t = blockDim.x / 2; t > 0; t /= 2) {
        if (threadIdx.x < t && local_mins[threadIdx.x + t] < local_mins[threadIdx.x]) {
            local_mins[threadIdx.x] = local_mins[threadIdx.x + t];
            local_mins_index[threadIdx.x] = local_mins_index[threadIdx.x + t];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        d_local_min[blockIdx.x] = local_mins[0];
        d_local_min_index[blockIdx.x] = local_mins_index[0];
    }
}

/*
 * function: update the global minimum value(and index), store them to a global memory address
 * parameters: *global_min: memory address to store the global min value, *global_min_index: memory address to store the global min index
 *        *d_local_min: array stores the local min value od each block, *d_local_min_index: array stores the local min index of each block
 *        *d_visit: array stores the status(visited/un-visited) for each vertex
 */
__global__ void
UpdateGlobalMin(int *global_min, int *global_min_index, int *d_local_min, int *d_local_min_index, int *d_visit) {
    int min = INF;
    int index = -1;

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < gridDim.x; i++) {
            if (d_local_min[i] < min) {
                min = d_local_min[i];
                index = d_local_min_index[i];
            }
        }
        global_min[0] = min;
        global_min_index[0] = index;
        d_visit[index] = 1;

    }
}

/*
 * function: update the shortest path for every un-visited vertices
 * parameters: N: input size, *mat: input matrix, *d_visit: array stores the status(visited/un-visited) for each vertex
 *             *d_all_dist: array stores the shortest distance for each vertex, *d_all_pred: array stores the predecessors
 *             *global_min: memory address that stores the global min value, *global_min_index: memory address that stores the global min index
 */
__global__ void
UpdatePath(int N, int *mat, int *d_visit, int *d_all_dist, int *d_all_pred, int *global_min, int *global_min_index) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    int g_min = global_min[0];
    int g_min_index = global_min_index[0];

    for (int i = tid; i < N; i += elementSkip) {
        int global_vertices_id = i;
        if (d_visit[global_vertices_id] == 0) {
            int new_dis = g_min + mat[g_min_index * N + global_vertices_id];
            if (new_dis < d_all_dist[global_vertices_id]) {
                d_all_dist[global_vertices_id] = new_dis;
                d_all_pred[global_vertices_id] = g_min_index;
            }
        }
    }
}

//Do not change anything below this line
void dijkstra(int N, int p, int *mat, int *all_dist, int *all_pred) {

    //threads number for each block should smaller than or equal to 1024
    assert(p <= 1024);

    //we restrict this value to 8, DO NOT change it!
    int blocksPerGrid = 8;

    //NOTICE: (p * 8) may LESS THAN N
    int threadsPerBlock = p;

    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);


    //allocate memory
    int *h_visit;
    int *d_mat, *d_visit, *d_all_dist, *d_all_pred, *d_local_min, *d_local_min_index;
    int *d_global_min, *d_global_min_index;

    h_visit = (int *) calloc(N, sizeof(int));
    hipMalloc(&d_mat, sizeof(int) * N * N);
    hipMalloc(&d_visit, sizeof(int) * N);
    hipMalloc(&d_all_dist, sizeof(int) * N);
    hipMalloc(&d_all_pred, sizeof(int) * N);
    hipMalloc(&d_local_min, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_local_min_index, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_global_min, sizeof(int));
    hipMalloc(&d_global_min_index, sizeof(int));

    //initialization and copy data from host to device
    for (int i = 0; i < N; i++) {
        all_dist[i] = mat[i];
        all_pred[i] = 0;
        h_visit[i] = 0;
    }
    h_visit[0] = 1;

    hipMemcpy(d_mat, mat, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_dist, all_dist, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_pred, all_pred, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_visit, h_visit, sizeof(int) * N, hipMemcpyHostToDevice);

    //dijkstra iterations
    for (int iter = 1; iter < N; iter++) {
        FindLocalMin << < blocks, threads >> > (N, d_visit, d_all_dist, d_local_min, d_local_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdateGlobalMin << < blocks, threads >> >
                                     (d_global_min, d_global_min_index, d_local_min, d_local_min_index, d_visit);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdatePath << < blocks, threads >> >
                                (N, d_mat, d_visit, d_all_dist, d_all_pred, d_global_min, d_global_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
    }

    //copy results from device to host
    hipMemcpy(all_dist, d_all_dist, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(all_pred, d_all_pred, sizeof(int) * N, hipMemcpyDeviceToHost);

    //free memory
    free(h_visit);
    hipFree(d_mat);
    hipFree(d_visit);
    hipFree(d_all_dist);
    hipFree(d_all_pred);
    hipFree(d_local_min);
    hipFree(d_local_min_index);
    hipFree(d_global_min);
    hipFree(d_global_min_index);

}

int main(int argc, char **argv) {
    assert(utils::parse_args(argc, argv) == 0);
    assert(utils::read_file(utils::filename) == 0);

    //`all_dist` stores the distances and `all_pred` stores the predecessors
    int *all_dist;
    int *all_pred;
    all_dist = (int *) calloc(utils::N, sizeof(int));
    all_pred = (int *) calloc(utils::N, sizeof(int));

    //time counter
    timeval start_wall_time_t, end_wall_time_t;
    float ms_wall;

    hipDeviceReset();

    //start timer
    gettimeofday(&start_wall_time_t, nullptr);
    dijkstra(utils::N, utils::num_threads, utils::mat, all_dist, all_pred);
    CHECK(hipDeviceSynchronize());

    //end timer
    gettimeofday(&end_wall_time_t, nullptr);
    ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
               + end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

    std::cerr << "Time(ms): " << ms_wall << endl;

    utils::print_result(all_dist, all_pred);

    free(utils::mat);
    free(all_dist);
    free(all_pred);

    return 0;
}
