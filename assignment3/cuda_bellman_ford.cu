#include "hip/hip_runtime.h"
/**
 * Name: Mu Cong DING
 * Student id: 20323458
 * ITSC email: mcding@connect.ust.hk
 */
/*
 * This is a CUDA version of bellman_ford algorithm
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
 * */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;

#define INF 1000000


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
		{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                               \
	}                                                                          \
		}


/**
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
 */
namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
	std::cerr << msg << endl;
	abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
	return x * n + y;
}

int read_file(string filename) {
	std::ifstream inputf(filename, std::ifstream::in);
	if (!inputf.good()) {
		abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
	}
	inputf >> N;
	//input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
	assert(N < (1024 * 1024 * 20));
	mat = (int *) malloc(N * N * sizeof(int));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++) {
			inputf >> mat[convert_dimension_2D_1D(i, j, N)];
		}
	return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
	std::ofstream outputf("output.txt", std::ofstream::out);
	if (!has_negative_cycle) {
		for (int i = 0; i < N; i++) {
			if (dist[i] > INF)
				dist[i] = INF;
			outputf << dist[i] << '\n';
		}
		outputf.flush();
	} else {
		outputf << "FOUND NEGATIVE CYCLE!" << endl;
	}
	outputf.close();
	return 0;
}
}//namespace utils

// you may add some helper/kernel functions here.

/**
 * function: BellmanIteration
 */
__global__ void BellmanIteration(int *d_n, int *d_mat, int *d_dist, bool *d_has_change, int *d_test) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int elementSkip = blockDim.x * gridDim.x;

    int n = *d_n;
	printf("LOG: %d, %d, %d\n", tid, elementSkip, n);
    *d_test = *d_test + 1;

	for (int i = tid; i < n * n; i += elementSkip) {
		int weight = d_mat[i];
		int u = i / n;
		int v = i - n * u;
		if (weight < 1000000) {//test if u--v has an edge
			if (d_dist[u] + weight < d_dist[v]) {
				*d_has_change = true;
				d_dist[v] = d_dist[u] + weight;
			}
		}
	}
}

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
	//------your code starts from here------

	//assert config parameters
	assert(4<= blocksPerGrid && blocksPerGrid <=32);
	assert(32<= threadsPerBlock && threadsPerBlock <= 1024);

	dim3 blocks(blocksPerGrid);
	dim3 threads(threadsPerBlock);

	//allocate memory
    int *d_n;
	int *d_mat, *d_dist;
    bool *d_has_change, *d_has_negative_cycle;
    int *d_test;
    gpuErrchk(hipMalloc(&d_n, sizeof(int)));
	gpuErrchk(hipMalloc(&d_mat, sizeof(int) * n * n));
	gpuErrchk(hipMalloc(&d_dist, sizeof(int) * n));
    gpuErrchk(hipMalloc(&d_has_change, sizeof(bool)));
    gpuErrchk(hipMalloc(&d_has_negative_cycle, sizeof(bool)));
    gpuErrchk(hipMalloc(&d_test, sizeof(int)));

	//initialization and copy data from host to device
	for (int i = 0; i < n; i++) {
		dist[i] = INF;
	}
	//root vertex always has distance 0
	dist[0] = 0;

	gpuErrchk(hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_mat, mat, sizeof(int) * n * n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_dist, dist, sizeof(int) * n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(d_has_change, 0, sizeof(bool)));
	gpuErrchk(hipMemset(d_has_negative_cycle, 0, sizeof(bool)));
	gpuErrchk(hipMemset(d_test, 0, sizeof(int)));


	//bellman-ford edge relaxation
	for (int i = 0; i < n - 1; i++) {// n - 1 iteration
		BellmanIteration << < blocks, threads >> > (d_n, d_mat, d_dist, d_has_change, d_test);
		gpuErrchk(hipDeviceSynchronize()); //only for debug
		int test;
        gpuErrchk(hipMemcpy(&test, d_test, sizeof(bool), hipMemcpyDeviceToHost));
		printf("%d\n", test);
	}

	//copy results from device to host
	gpuErrchk(hipMemcpy(dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(has_negative_cycle, d_has_negative_cycle, sizeof(bool), hipMemcpyDeviceToHost));

	//free memory
	gpuErrchk(hipFree(d_mat));
	gpuErrchk(hipFree(d_dist));
	//------end of your code------
}

int main(int argc, char **argv) {
	if (argc <= 1) {
		utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
	}
	if (argc <= 3) {
		utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
	}

	string filename = argv[1];
	int blockPerGrid = atoi(argv[2]);
	int threadsPerBlock = atoi(argv[3]);

	int *dist;
	bool has_negative_cycle = false;


	assert(utils::read_file(filename) == 0);
	dist = (int *) calloc(sizeof(int), utils::N);


	//time counter
	timeval start_wall_time_t, end_wall_time_t;
	float ms_wall;
	hipDeviceReset();
	//start timer
	gettimeofday(&start_wall_time_t, nullptr);
	//bellman-ford algorithm
	bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
	CHECK(hipDeviceSynchronize());
	//end timer
	gettimeofday(&end_wall_time_t, nullptr);
	ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
			+ end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

	std::cerr.setf(std::ios::fixed);
	std::cerr << std::setprecision(6) << "Time(s): " << (ms_wall/1000.0) << endl;
	utils::print_result(has_negative_cycle, dist);
	free(dist);
	free(utils::mat);

	return 0;
}
